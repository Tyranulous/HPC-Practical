#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand.h>

#define CUDA_CALL(x) do { if((x)!=hipSuccess) {\
	printf("Error at %s:%d\n", __FILE__,__LINE__);\
	return EXIT_FAILURE;}} while(0)

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
	printf("Error");\
	return EXIT_FAILURE;}} while(0)

int main(void)
{
	
	size_t n = 100;
	size_t i;
	hiprandGenerator_t gen;
	float *d_data, *h_data;
	float mean, stddev;

//Set mean and standard deviation
	mean = 0.0;
	stddev = 1.0;

// Allocate n floats on host
	
	h_data = (float *)malloc(n*sizeof(h_data));
	

// Allocate n floats on device

	CUDA_CALL(hipMalloc((void **)&d_data, n*sizeof(d_data)));

//Create random number generator
	CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));

//Set seed
	CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));

//Generate n normally distributed values
	CURAND_CALL(hiprandGenerateNormal(gen, d_data, n, mean, stddev));

//Copy generated numbers to host
	CUDA_CALL(hipMemcpy(h_data, d_data,n * sizeof(float), hipMemcpyDeviceToHost));

//Print results
	for (i = 0; i < n; i++)
	{
		printf("%1.4f ", h_data[i]);
		printf("\n");
 	}

//Cleanup
	CURAND_CALL(hiprandDestroyGenerator(gen));
	CUDA_CALL(hipFree(d_data));
	free(h_data);
	return EXIT_SUCCESS;
}

